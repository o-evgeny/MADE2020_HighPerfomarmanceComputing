#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

// kernel
__global__ void __apply(const uint8_t *pic, uint32_t *histogram) {
  int32_t i = blockIdx.x;
  int32_t j = threadIdx.x;
  int32_t n_col = blockDim.x;
  int32_t n_row = gridDim.x;
  uint32_t pixel_current = i * n_col + j;
  __syncthreads();
  atomicAdd(histogram+pic[pixel_current],1);
  //histogram[pic[pixel_current]]+=1;
  __syncthreads();
}

__host__ void histogram_grayscale(const uint8_t *h_pic, uint32_t *h_histogram, size_t n_row, size_t n_col) {
  // перенос картинки и вывода
  size_t size = sizeof(uint8_t) * n_row * n_col;
  uint8_t *d_pic;
  hipMalloc(&d_pic, size);
  hipMemcpy(d_pic, h_pic, size, hipMemcpyHostToDevice);
  size_t size_hist = sizeof(uint32_t)*256;
  uint32_t* d_histogram;
  hipMalloc(&d_histogram, size_hist);
  hipMemset(d_histogram, 0, size_hist);


  //__apply<<<dim3(n_row, 1, 1), dim3(n_col, 1, 1)>>>(d_pic, d_pic_filter, d_filter, sum_filter, n_row_filter, n_col_filter);
  __apply<<<dim3(n_row, 1, 1), dim3(n_col, 1, 1)>>>(d_pic, d_histogram);

  if (n_col>1024)
	printf("Big picture. Remake separate threads. Not build histogram");
  hipMemcpy(h_histogram, d_histogram, size_hist, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(d_pic);
  hipFree(d_histogram);
}