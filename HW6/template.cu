#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

// kernel
__global__ void __apply(const uint8_t *pic, uint8_t *pic_filter,
						double *filter, double sum_filter, int32_t n_row_filter, int32_t n_col_filter) {
  int32_t i = blockIdx.x;
  int32_t j = threadIdx.x;
  int32_t n_col = blockDim.x;
  int32_t n_row = gridDim.x;
  uint32_t pixel_current = i * n_col + j;
  // края не размываю
  if (((i - n_row_filter) < 0) |
	  ((i + n_row_filter+2) > n_row) |
	  ((j - n_col_filter) < 0) |
	  ((j + n_col_filter+2) > n_col)) {
	pic_filter[pixel_current] = pic[pixel_current];
	return;
  }
  pic_filter[pixel_current]=0;
  double filt;
  uint8_t pic_part;

  for (int k = 0; k < n_row_filter; ++k) {
	for (int l = 0; l < n_col_filter; ++l) {
	  filt = filter[k*5 + l];
	  pic_part = pic[(i+k-n_row_filter/2)*n_col + (j+l-n_col_filter/2)];
	  //pic_part = pic[i * n_col + j];
	  //printf("%f %f %i k=%i l=%i n_row_filter=%i n_col=%i\n",sum_filter, filt, pic_part, k, l, n_row_filter, n_col);
	  pic_filter[pixel_current] += 1.0/sum_filter*filt*pic_part;
	}
	}
  __syncthreads();
  }

  __host__ void filter_template_grayscale(const uint8_t *h_pic, uint8_t *h_pic_filter, size_t n_row, size_t n_col) {

	// задание шаблона
	  int32_t n_row_filter = 3;
	  int32_t n_col_filter = 3;
	  double sum_filter = 0.0;
	  double h_filter[9];
	  for (auto &f : h_filter) {
		f = 0.9;
		sum_filter += f;
	  }
	double *d_filter;
	hipMalloc(&d_filter, sizeof(double) * n_row_filter * n_col_filter);
	hipMemcpy(d_filter, h_filter, sizeof(double) * n_row_filter * n_col_filter, hipMemcpyHostToDevice);


	// перенос картинки и вывода
	size_t size = sizeof(uint8_t) * n_row * n_col;
	uint8_t *d_pic;
	hipMalloc(&d_pic, size);
	hipMemcpy(d_pic, h_pic, size, hipMemcpyHostToDevice);
	uint8_t *d_pic_filter;
	hipMalloc(&d_pic_filter, size);
	hipMemset(d_pic_filter, 0, size);


	//__apply<<<dim3(n_row, 1, 1), dim3(n_col, 1, 1)>>>(d_pic, d_pic_filter, d_filter, sum_filter, n_row_filter, n_col_filter);
	__apply<<<dim3(n_row, 1, 1), dim3(n_col, 1, 1)>>>(d_pic, d_pic_filter, d_filter, sum_filter, n_row_filter, n_col_filter);

	if (n_col>1024)
	  printf("Big picture. Remake separate threads. Picture not filtered");
	hipMemcpy(h_pic_filter, d_pic_filter, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_pic);
	hipFree(d_pic_filter);
  }