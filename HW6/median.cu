#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

// kernel
__global__ void __apply(const uint8_t *pic, uint8_t *pic_filter, int32_t n_row_window, int32_t n_col_window) {
  int32_t i = blockIdx.x;
  int32_t j = threadIdx.x;
  int32_t n_col = blockDim.x;
  int32_t n_row = gridDim.x;
  uint32_t pixel_current = i * n_col + j;
  // края не размываю
  if (((i - n_row_window) < 0) |
	  ((i + n_row_window+2) > n_row) |
	  ((j - n_col_window) < 0) |
	  ((j + n_col_window+2) > n_col)) {
	pic_filter[pixel_current] = pic[pixel_current];
	return;
  }
  //median от window
  uint32_t map_pixel_to_count[256];
  memset(map_pixel_to_count, 0 ,sizeof(uint32_t)*256);
  for(int window_row=0;window_row<n_row_window;++window_row)
	for (int window_col=0;window_col<n_col_window;++window_col){
	  map_pixel_to_count[pic[(i+window_row-n_row_window/2)*n_col+(j+window_col-n_row_window/2)]]+=1;
	}
  uint32_t tmp=0;
  uint8_t pixel_median;
  uint32_t half_count_element_window = (n_row_window*n_col_window)/2;
  for(pixel_median=0;pixel_median!=255;++pixel_median){
	tmp+=map_pixel_to_count[pixel_median];
	if(tmp>half_count_element_window)
		break;
  }
  pic_filter[pixel_current]=pixel_median;
  __syncthreads();
}

__host__ void filter_median_grayscale(const uint8_t *h_pic, uint8_t *h_pic_filter, size_t n_row, size_t n_col) {
  // задание шаблона
  int32_t n_row_filter = 3;
  int32_t n_col_filter = 3;

  // перенос картинки и вывода
  size_t size = sizeof(uint8_t) * n_row * n_col;
  uint8_t *d_pic;
  hipMalloc(&d_pic, size);
  hipMemcpy(d_pic, h_pic, size, hipMemcpyHostToDevice);
  uint8_t *d_pic_filter;
  hipMalloc(&d_pic_filter, size);
  hipMemset(d_pic_filter, 0, size);


  //__apply<<<dim3(n_row, 1, 1), dim3(n_col, 1, 1)>>>(d_pic, d_pic_filter, d_filter, sum_filter, n_row_filter, n_col_filter);
  __apply<<<dim3(n_row, 1, 1), dim3(n_col, 1, 1)>>>(d_pic, d_pic_filter, n_row_filter, n_col_filter);

  if (n_col>1024)
	printf("Big picture. Remake separate threads. Picture not filtered");
  hipMemcpy(h_pic_filter, d_pic_filter, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(d_pic);
  hipFree(d_pic_filter);
}